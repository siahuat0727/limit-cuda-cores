#include <iostream>
#include <vector>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <unistd.h>
#include <assert.h>

using namespace std;
using clock_value_t = long long;

static __device__ __inline__ uint32_t __mysmid() {
    uint32_t smid;
    asm volatile("mov.u32 %0, %%smid;" : "=r"(smid));
    return smid;
}

__device__ void sleepForever()
{
    clock_value_t sleep_cycles = 800000000000LL;  // TODO can we sleep forever?
    clock_value_t start = clock64();
    clock_value_t cycles_elapsed;
    do { 
        cycles_elapsed = clock64() - start; 
    } while (cycles_elapsed < sleep_cycles);

    printf("Never reach here!");
}

__device__ void smSleep(clock_value_t sleep_cycles)
{
    clock_value_t start = clock64();
    clock_value_t cycles_elapsed;
    do { 
        cycles_elapsed = clock64() - start; 
    } while (cycles_elapsed < sleep_cycles);
}

__global__ void sleepKernel(int target_cores_num) {
    uint32_t smid = __mysmid();
    if (smid >= target_cores_num) {
        printf("My SM ID is %d, sleep forever\n", smid);
        sleepForever();
    } else {
        printf("My SM ID is %d, take a snap about 2 s\n", smid);
        smSleep(5000000000LL);  // TODO can convert to seconds?
    }
    printf("My SM ID is %d, wake up!\n", smid);
}

__global__ void helloWorldKernel() {
    uint32_t smid = __mysmid();
    printf("Hello world! My SM ID is %d\n", smid);
}

void doLimitSM(int target_cores_num, hipStream_t stream){
    dim3 threadsPerBlock(1, 1);
    dim3 blocksPerGrid(68, 1);  // TODO get this automatically
    sleepKernel<<<blocksPerGrid,threadsPerBlock, 0, stream>>>(target_cores_num);
}

void limitSM(int target_cores_num) {
    int max_stream = 16; // TODO get this automatically
    hipStream_t stream[max_stream];
    for (int i = 0; i < max_stream; ++i) {
        hipStreamCreate(&stream[i]);
        doLimitSM(target_cores_num, stream[i]);
    }
}

void helloWorld(hipStream_t stream){
    dim3 threadsPerBlock(1, 1);
    dim3 blocksPerGrid(68, 1);
    helloWorldKernel<<<blocksPerGrid,threadsPerBlock, 0, stream>>>();
}

void mainTask() {
    hipStream_t stream;
    hipStreamCreate(&stream);

    helloWorld(stream);

    hipStreamSynchronize(stream);
}

int main()
{
    int target_cores_num = 2;
    limitSM(target_cores_num);

    sleep(5);

    puts("-----");
    puts("Main task start");
    mainTask();
    puts("Main task end");

    return 0;
}
